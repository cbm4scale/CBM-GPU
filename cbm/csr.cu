#include <torch/extension.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
    }                                                                          \
}

class cusparseCSR {
public:
    cusparseCSR(const torch::Tensor& adjacency) {

        // keep a copy 
        tensor_ = &adjacency;

        // init csr metadata
        nnz_ = adjacency.values().size(0);
        num_rows_ = adjacency.size(0);
        num_cols_ = adjacency.size(1);

        // init csr arrays
        row_ptr_ = adjacency.crow_indices().data_ptr<int32_t>();
        col_idx_ = adjacency.col_indices().data_ptr<int32_t>();
        values_ = adjacency.values().data_ptr<float>();

        // init cusparse
        CHECK_CUSPARSE( hipsparseCreate(&handle_) )
        
        // init csr matrix with cusparse
        CHECK_CUSPARSE( hipsparseCreateCsr(&descr_, num_rows_, num_cols_, nnz_,
                                          row_ptr_, col_idx_, values_,
                                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    }

    ~cusparseCSR() {
        CHECK_CUSPARSE( hipsparseDestroy(handle_) )
        CHECK_CUSPARSE( hipsparseDestroySpMat(descr_) )
        printf("destroying csr instance on GPU...\n");
    }

    int matmul(const torch::Tensor& rhs, torch::Tensor& res) {
        TORCH_CHECK(rhs.is_cuda(), "tensor is not a CUDA tensor");
        TORCH_CHECK(res.is_cuda(), "tensor is not a CUDA tensor");
        
        float *rhs_ptr = rhs.data_ptr<float>();
        float *res_ptr = res.data_ptr<float>();
        
        hipsparseDnMatDescr_t rhs_descr;
        CHECK_CUSPARSE (
            hipsparseCreateDnMat(&rhs_descr, num_cols_, rhs.size(1), rhs.size(1), 
                                (void *)rhs_ptr, HIP_R_32F, HIPSPARSE_ORDER_ROW)
        )
        
        hipsparseDnMatDescr_t res_descr;
        CHECK_CUSPARSE (
            hipsparseCreateDnMat(&res_descr, num_rows_, res.size(1), res.size(1), 
                                (void *)res_ptr, HIP_R_32F, HIPSPARSE_ORDER_ROW)
        )

        size_t bufferSize = 0;
        void *dBuffer = nullptr;
        float alpha = 1.0f, beta = 0.0f;
        CHECK_CUSPARSE (
            hipsparseSpMM_bufferSize(handle_, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, 
                                    descr_, rhs_descr, &beta, res_descr, 
                                    HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, 
                                    &bufferSize)
        )

        // Allocate buffer
        CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

        // Perform sparse-dense matrix multiplication
        CHECK_CUSPARSE (
            hipsparseSpMM(handle_, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, 
                        descr_, rhs_descr, &beta, res_descr, 
                        HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer)
        )

        // Free resources
        CHECK_CUDA( hipFree(dBuffer) )
        CHECK_CUSPARSE ( hipsparseDestroyDnMat(rhs_descr) )
        CHECK_CUSPARSE ( hipsparseDestroyDnMat(res_descr) )
        return 0;
    }

private:
    // keep tensor alive (is this really needed?)
    const torch::Tensor *tensor_; 

    // csr matrix metadata 
    int32_t nnz_; 
    int32_t num_rows_; 
    int32_t num_cols_;
    
    // deltas(csr) matrix coordinates
    int32_t *row_ptr_; 
    int32_t *col_idx_;

    // deltas (csr) matrix values
    float *values_;

    // cusparse handle
    hipsparseHandle_t handle_;

    // cusparse csr descriptor
    hipsparseSpMatDescr_t descr_;
};

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    py::class_<cusparseCSR>(m, "cusparseCSR")
        .def(py::init<
                const torch::Tensor&>())
        .def("matmul", &cusparseCSR::matmul);
}